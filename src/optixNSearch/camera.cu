#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <hip/hip_vector_types.h>
#include <optix_device.h>

#include "optixNSearch.h"
#include "helpers.h"

extern "C" {
__constant__ Params params;
}

extern "C" __global__ void __raygen__knn()
{
    const uint3 idx = optixGetLaunchIndex();
    unsigned int rayIdx = idx.x;

    // if d_r2q_map is null, it could be 1) an unsorted run, 2) an initial run
    // for sorting, or 3) a sorted run with queries pre-gathered. Either case,
    // we directly map rays to queries.

    unsigned int queryIdx;
    if (params.d_r2q_map == nullptr)
      queryIdx = rayIdx;
    else
      queryIdx = params.d_r2q_map[rayIdx];

    float3 ray_origin = params.queries[queryIdx];
    float3 ray_direction = normalize(make_float3(1, 0, 0));

    //if (queryIdx == 163455) {
    //  printf("%u, %u!!!\n", queryIdx, rayIdx);
    //  printf("%f, %f, %f!!!\n", ray_origin.x, ray_origin.y, ray_origin.z);
    //}

    const float tmin = 0.f;
    const float tmax = 1.e-16f;

    // pointers are 64 bits, so need two 32-bit integers. optixPathTracing has an example for this.
    float min_dists[K];
    unsigned int u0, u1;
    packPointer( min_dists, u0, u1 );

    unsigned int min_idxs[K];
    unsigned int u2, u3;
    packPointer( min_idxs, u2, u3 );

    float max_key;
    unsigned int max_idx;
    unsigned int size = 0;

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        RAY_TYPE_RADIANCE,
        1,
        RAY_TYPE_RADIANCE,
        reinterpret_cast<unsigned int&>(queryIdx),
        u0, u1, // min_dists
        u2, u3, // min_idxs
        reinterpret_cast<unsigned int&>(max_key),
        reinterpret_cast<unsigned int&>(max_idx),
        reinterpret_cast<unsigned int&>(size)
    );

    // write minK queue data to frame_buffer if is an actual search, i.e., not initial traversal
    if (params.mode == PRECISE) { // implies this is an actual search
      // the bound should be |size| rather than K (size <= K) so that we don't have to initialize min_idxs!
      for (unsigned int i = 0; i < size; i++) {
        params.frame_buffer[queryIdx * K + i] = min_idxs[i];
      }
    }
}

extern "C" __global__ void __raygen__radius()
{
    const uint3 idx = optixGetLaunchIndex();
    unsigned long long rayIdx = idx.x;

    // if d_r2q_map is null, it could be 1) an unsorted run, 2) an initial run
    // for sorting, or 3) a sorted run with queries pre-gathered. Either case,
    // we directly map rays to queries.

    unsigned int queryIdx;
    if (params.d_r2q_map == nullptr)
      queryIdx = rayIdx;
    else
      queryIdx = params.d_r2q_map[rayIdx];

    float3 ray_origin = params.queries[queryIdx];
    float3 ray_direction = normalize(make_float3(1, 0, 0));

    unsigned int id = 0;
    const float tmin = 0.f;
    const float tmax = 1.e-16f;

    optixTrace(
        params.handle,
        ray_origin,
        ray_direction,
        tmin,
        tmax,
        0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        //OPTIX_RAY_FLAG_DISABLE_ANYHIT |
        //OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT,
        RAY_TYPE_RADIANCE,
        1,
        RAY_TYPE_RADIANCE,
        reinterpret_cast<unsigned int&>(queryIdx),
        reinterpret_cast<unsigned int&>(id)
    );
}
